#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <gl/freeglut.h>
#include <stdio.h>
#include <omp.h>
#include <math.h>

#define WIDTH 800
#define HEIGHT 800
#define ITERATIONS 5000


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

typedef struct rgb_color {
	float r;
	float g;
	float b;
}color;

color colors[16];
color pixels[WIDTH*HEIGHT];

double real_min = -2, real_max = 1;
double img_min = -1.5, img_max = 1.5;
int MAX_ITER = ITERATIONS; int iter_step = 10; 

int window;

int size = HEIGHT * WIDTH * sizeof(int); //device aloc size
int *iters = (int *)malloc(size);
int *iters_d;

int arr_size = WIDTH * HEIGHT;
int num_threads = 512;
int block_size = 1250;
double t_start, t_end;

void initColors() {
	colors[0].r = 66; colors[0].g = 30; colors[0].b = 15;
	colors[1].r = 25; colors[1].g = 7; colors[1].b = 26;
	colors[2].r = 9; colors[2].g = 1; colors[2].b = 47;
	colors[3].r = 4; colors[3].g = 4; colors[3].b = 73;
	colors[4].r = 0; colors[4].g = 7; colors[4].b = 100;
	colors[5].r = 12; colors[5].g = 44; colors[5].b = 138;
	colors[6].r = 24; colors[6].g = 82; colors[6].b = 177;
	colors[7].r = 57; colors[7].g = 125; colors[7].b = 209;
	colors[8].r = 134; colors[8].g = 181; colors[8].b = 229;
	colors[9].r = 211; colors[9].g = 236; colors[9].b = 248;
	colors[10].r = 241; colors[10].g = 233; colors[10].b = 191;
	colors[11].r = 248; colors[11].g = 201; colors[11].b = 95;
	colors[12].r = 255; colors[12].g = 170; colors[12].b = 0;
	colors[13].r = 204; colors[13].g = 128; colors[13].b = 0;
	colors[14].r = 153; colors[14].g = 87; colors[14].b = 0;
	colors[15].r = 106; colors[15].g = 52; colors[15].b = 3;
}

void setUpColor() {
	const int NX = WIDTH;
	const int NY = HEIGHT;
	int i, j, VAL;

	for (i = 0; i < NX; i++) {
		for (j = 0; j < NY; j++) {
			VAL = iters[i*WIDTH + j];

			if (VAL < MAX_ITER && VAL > 0) {
				int cid = VAL % 16;

				pixels[i + j*HEIGHT].r = colors[cid].r / 255;
				pixels[i + j*HEIGHT].g = colors[cid].g / 255;
				pixels[i + j*HEIGHT].b = colors[cid].b / 255;
			}
			else {
				pixels[i + j*HEIGHT].r = 0;
				pixels[i + j*HEIGHT].g = 0;
				pixels[i + j*HEIGHT].b = 0;
			}
		}
	}
}

__global__ void mandelbrotset(int *iter_arr, double xmin, double xmax, double ymin, double ymax, int w, int h, int max_iters, int N) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index >= N)
		return;

	const int NX = w;
	const int NY = h;

	double dx = (xmax - xmin) / NX;
	double dy = (ymax - ymin) / NY;

	int i = index / w;
	int j = index % h;

	double x = xmin + i*dx;
	double y = ymin + j*dy;

	double c_i = y;
	double c_r = x;
	double z_r = 0, z_i = 0;
	int n;

	for ( n = 0; n < max_iters; ++n) {
		double z_r2 = z_r*z_r, z_i2 = z_i*z_i;
		if (z_r2 + z_i2 > 4) {
			break;
		}
		z_i = 2 * z_r*z_i + c_i;
		z_r = z_r2 - z_i2 + c_r;
	}

	iter_arr[index] = n;
}

void SpecialKeys(int key, int x, int y) {

	double xStep = fabs(real_min - real_max) * 0.08;
	double yStep = fabs(img_min - img_max) * 0.08;


	switch (key) {
	case GLUT_KEY_LEFT:
		real_min -= xStep;
		real_max -= xStep;
		break;

	case GLUT_KEY_RIGHT:
		real_min += xStep;
		real_max += xStep;
		break;

	case GLUT_KEY_UP:
		img_min += yStep;
		img_max += yStep;
		break;

	case GLUT_KEY_DOWN:
		img_min -= yStep;
		img_max -= yStep;
		break;
	}
	glutPostRedisplay();
}

void KeyB(unsigned char key, int x, int y) {
	double xStep = fabs(real_min - real_max) * 0.08;
	double yStep = fabs(img_min - img_max) * 0.08;


	switch (key) {
	case '+':
		real_min += xStep * 2;
		real_max -= xStep * 2;
		img_min += yStep * 2;
		img_max -= yStep * 2;
		break;

	case '-':
		real_min -= xStep * 2;
		real_max += xStep * 2;
		img_min -= yStep * 2;
		img_max += yStep * 2;
		break;

	case 27: // Escape key
		delete(iters);
		glutDestroyWindow(window);
		hipFree(iters_d);
		exit(0);
		break;

	case 'r':
		MAX_ITER += iter_step;
		printf("Iterations: %d -> %d\n", MAX_ITER - 10, MAX_ITER);
		break;

	case 't':
		MAX_ITER -= iter_step;
		printf("Iterations: %d -> %d\n", MAX_ITER + 10, MAX_ITER);
		break;
	}
	glutPostRedisplay();
}

void onDisplay() {
	t_start = omp_get_wtime();

	mandelbrotset<<<(block_size*num_threads + num_threads - 1) / num_threads,num_threads>>>(iters_d, real_min, real_max, img_min, img_max, WIDTH, HEIGHT, MAX_ITER, arr_size);
	
	gpuErrchk(hipMemcpy(iters, iters_d, size, hipMemcpyDeviceToHost));
	
	t_end = omp_get_wtime();
	printf("Render time: %lf\n", t_end - t_start);
	
	setUpColor();

	glClearColor(1, 1, 1, 0);
	glClear(GL_COLOR_BUFFER_BIT);

	glDrawPixels(WIDTH, HEIGHT, GL_RGB, GL_FLOAT, pixels);
	glutSwapBuffers();
}

void Init() {
	glutInitWindowSize(WIDTH, HEIGHT);
	glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
	glutInitWindowPosition(100, 100);
	window = glutCreateWindow("Mandelbrotset");

	glutKeyboardFunc(KeyB);
	glutSpecialFunc(SpecialKeys);
	glViewport(0, 0, HEIGHT, WIDTH);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0, WIDTH, 0, HEIGHT);


	gpuErrchk(hipMalloc((void **)&iters_d, size));

	initColors();
}

int main(int argc, char** argv) {
		glutInit(&argc, argv);
		Init();

		glutDisplayFunc(onDisplay);
		glutMainLoop();

	return 0;
}
